#include "hip/hip_runtime.h"
//---------------------------------*-CUDA-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file KNDemoKernel.cu
//---------------------------------------------------------------------------//
#include "KNDemoKernel.hh"

#include <alpaka/alpaka.hpp>

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include "base/ArrayUtils.hh"
#include "base/Assert.hh"
#include "physics/base/ParticleTrackView.hh"
#include "physics/base/SecondaryAllocatorView.hh"
#include "physics/em/KleinNishinaInteractor.hh"
#include "random/cuda/RngEngine.cuh"
#include "random/distributions/ExponentialDistribution.hh"
#include "PhysicsArrayCalculator.hh"
#include "DetectorView.hh"

using namespace celeritas;

namespace demo_interactor
{
//---------------------------------------------------------------------------//
/*!
 * Kernel to initialize particle data.
 *
 * For testing purposes (this might not be the case for the final app) we use a
 * grid-stride loop rather than requiring that each thread correspond exactly
 * to a particle track. In other words, this method allows a single warp to
 * operate on two 32-thread chunks of data.
 *  https://developer.nvidia.com/blog/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
 */
__global__ void initialize_kn(ParamPointers const   params,
                              StatePointers const   states,
                              InitialPointers const init)
{
    // Grid-stride loop, see
    for (int tid = blockIdx.x * blockDim.x + threadIdx.x;
         tid < static_cast<int>(states.size());
         tid += blockDim.x * gridDim.x)
    {
        ParticleTrackView particle(
            params.particle, states.particle, ThreadId(tid));
        particle = init.particle;

        // Particles begin alive and in the +z direction
        states.direction[tid] = {0, 0, 1};
        states.position[tid]  = {0, 0, 0};
        states.time[tid]      = 0;
        states.alive[tid]     = true;
    }
}


using namespace alpaka;

//Define shortcuts for some alpaka items we will use
using Dim = dim::DimInt<1>;
using Idx = uint32_t;
//Define the alpaka accelerator to be Nvidia GPU
using Acc = acc::AccGpuCudaRt<Dim,Idx>;

struct initialize_alpaka{
  template <typename Acc>
  ALPAKA_FN_ACC void operator()(Acc const &acc,ParamPointers const params,StatePointers const states,InitialPointers const init) const {
    for(int tid = idx::getIdx<Grid, Threads>(acc)[0];tid < static_cast<int>(states.size());tid += blockDim.x * gridDim.x){
      ParticleTrackView particle(params.particle, states.particle, ThreadId(tid));
      particle = init.particle;

      // Particles begin alive and in the +z direction
      states.direction[tid] = {0, 0, 1};
      states.position[tid]  = {0, 0, 0};
      states.time[tid]      = 0;
      states.alive[tid]     = true;
    }

  }
};


//---------------------------------------------------------------------------//
/*!
 * Perform a single interaction per particle track.
 *
 * The interaction:
 * - Clears the energy deposition
 * - Samples the KN interaction
 * - Allocates and emits a secondary
 * - Kills the secondary, depositing its local energy
 * - Applies the interaction (updating track direction and energy)
 */
__global__ void iterate_kn(ParamPointers const              params,
                           StatePointers const              states,
                           SecondaryAllocatorPointers const secondaries,
                           DetectorPointers const           detector)
{
    SecondaryAllocatorView allocate_secondaries(secondaries);
    DetectorView           detector_hit(detector);
    PhysicsArrayCalculator calc_xs(params.xs);

    for (int tid = blockIdx.x * blockDim.x + threadIdx.x;
         tid < static_cast<int>(states.size());
         tid += blockDim.x * gridDim.x)
    {
        // Skip loop if already dead
        if (!states.alive[tid])
        {
            continue;
        }

        // Construct particle accessor from immutable and thread-local data
        ParticleTrackView particle(
            params.particle, states.particle, ThreadId(tid));
        RngEngine rng(states.rng, ThreadId(tid));

        // Move to collision
        {
            // Calculate cross section at the particle's energy
            real_type                          sigma = calc_xs(particle);
            ExponentialDistribution<real_type> sample_distance(sigma);
            // Sample distance-to-collision
            real_type distance = sample_distance(rng);
            // Move particle
            axpy(distance, states.direction[tid], &states.position[tid]);
            // Update time
            states.time[tid] += distance * unit_cast(particle.speed());
        }

        Hit h;
        h.pos    = states.position[tid];
        h.thread = ThreadId(tid);
        h.time   = states.time[tid];

        if (particle.energy() < KleinNishinaInteractor::min_incident_energy())
        {
            // Particle is below interaction energy
            h.dir              = states.direction[tid];
            h.energy_deposited = particle.energy();

            // Deposit energy and kill
            detector_hit(h);
            states.alive[tid] = false;
            continue;
        }

        // Construct RNG and interaction interfaces
        KleinNishinaInteractor interact(params.kn_interactor,
                                        particle,
                                        states.direction[tid],
                                        allocate_secondaries);

        // Perform interaction: should emit a single particle (an electron)
        Interaction interaction = interact(rng);
        CHECK(interaction);
        CHECK(interaction.secondaries.size() == 1);

        // Deposit energy from the secondary (effectively, an infinite energy
        // cutoff)
        {
            const auto& secondary = interaction.secondaries.front();
            h.dir                 = secondary.direction;
            h.energy_deposited    = secondary.energy;
            detector_hit(h);
        }

        // Update post-interaction state (apply interaction)
        states.direction[tid] = interaction.direction;
        particle.energy(interaction.energy);
    }
}

struct iterate_alpaka{
  template <typename Acc>
  ALPAKA_FN_ACC void operator()(Acc const &acc,ParamPointers const params, StatePointers const states, SecondaryAllocatorPointers const secondaries, DetectorPointers const detector) const{

    SecondaryAllocatorView allocate_secondaries(secondaries);
    DetectorView           detector_hit(detector);
    PhysicsArrayCalculator calc_xs(params.xs);

    for(int tid = idx::getIdx<Grid, Threads>(acc)[0];tid < static_cast<int>(states.size());tid += blockDim.x * gridDim.x){

      // Skip loop if already dead
      if (!states.alive[tid])
      {
          continue;
      }

      // Construct particle accessor from immutable and thread-local data
      ParticleTrackView particle(
          params.particle, states.particle, ThreadId(tid));
      RngEngine rng(states.rng, ThreadId(tid));

      // Move to collision
      {
          // Calculate cross section at the particle's energy
          real_type                          sigma = calc_xs(particle);
          ExponentialDistribution<real_type> sample_distance(sigma);
          // Sample distance-to-collision
          real_type distance = sample_distance(rng);
          // Move particle
          axpy(distance, states.direction[tid], &states.position[tid]);
          // Update time
          states.time[tid] += distance * unit_cast(particle.speed());
      }

      Hit h;
      h.pos    = states.position[tid];
      h.thread = ThreadId(tid);
      h.time   = states.time[tid];

      if (particle.energy() < KleinNishinaInteractor::min_incident_energy())
      {
          // Particle is below interaction energy
          h.dir              = states.direction[tid];
          h.energy_deposited = particle.energy();

          // Deposit energy and kill
          detector_hit(h);
          states.alive[tid] = false;
          continue;
      }

      // Construct RNG and interaction interfaces
      KleinNishinaInteractor interact(params.kn_interactor,
                                      particle,
                                      states.direction[tid],
                                      allocate_secondaries);

      // Perform interaction: should emit a single particle (an electron)
      Interaction interaction = interact(rng);
      CHECK(interaction);
      CHECK(interaction.secondaries.size() == 1);

      // Deposit energy from the secondary (effectively, an infinite energy
      // cutoff)
      {
          const auto& secondary = interaction.secondaries.front();
          h.dir                 = secondary.direction;
          h.energy_deposited    = secondary.energy;
          detector_hit(h);
      }

      // Update post-interaction state (apply interaction)
      states.direction[tid] = interaction.direction;
      particle.energy(interaction.energy);

    }

  }
};

//---------------------------------------------------------------------------//
// HOST INTERFACES
//---------------------------------------------------------------------------//
/*!
 * Initialize particle states.
 */
void initialize(const CudaGridParams&  grid,
                const ParamPointers&   params,
                const StatePointers&   states,
                const InitialPointers& initial)
{
    REQUIRE(states.alive.size() == states.size());
    REQUIRE(states.rng.size() == states.size());
    //initialize_kn<<<grid.grid_size, grid.block_size>>>(params, states, initial);
  
    //Get the first device available of type GPU (i.e should be our sole GPU)/device
    auto const device = pltf::getDevByIdx<Acc>(0u);
    auto queue = queue::Queue<Acc, queue::Blocking>{device};
    auto workDiv = workdiv::WorkDivMembers<Dim, Idx>{static_cast<uint32_t>(grid.block_size), static_cast<uint32_t>(grid.grid_size), static_cast<uint32_t>(1)};

    //Create a task for processEvent, that we can run and then run it via a queue
    initialize_alpaka initialize_alpaka;
    auto taskInitialize = kernel::createTaskKernel<Acc>(workDiv,initialize_alpaka,params,states,initial);
    queue::enqueue(queue, taskInitialize);

}

//---------------------------------------------------------------------------//
/*!
 * Run an iteration.
 */
void iterate(const CudaGridParams&              grid,
             const ParamPointers&               params,
             const StatePointers&               state,
             const SecondaryAllocatorPointers&  secondaries,
             const celeritas::DetectorPointers& detector)
{
    //iterate_kn<<<grid.grid_size, grid.block_size>>>(
    //    params, state, secondaries, detector);

    // Note: the device synchronize is useful for debugging and necessary for
    // timing diagnostics.
    //CELER_CUDA_CALL(hipDeviceSynchronize());

    //Get the first device available of type GPU (i.e should be our sole GPU)/device
    auto const device = pltf::getDevByIdx<Acc>(0u);
    auto queue = queue::Queue<Acc, queue::Blocking>{device};
    auto workDiv = workdiv::WorkDivMembers<Dim, Idx>{static_cast<uint32_t>(grid.block_size), static_cast<uint32_t>(grid.grid_size), static_cast<uint32_t>(1)};

    //Create a task for iterate, that we can run and then run it via a queue
    iterate_alpaka iterate_alpaka;
    auto taskIterate = kernel::createTaskKernel<Acc>(workDiv,iterate_alpaka,params, state, secondaries, detector);
    queue::enqueue(queue, taskIterate);

}

//---------------------------------------------------------------------------//
/*!
 * Sum the total number of living particles.
 */
size_type reduce_alive(span<bool> alive)
{
    size_type result = thrust::reduce(
        thrust::device_pointer_cast(alive.data()),
        thrust::device_pointer_cast(alive.data() + alive.size()),
        size_type(0),
        thrust::plus<size_type>());

    CELER_CUDA_CALL(hipDeviceSynchronize());
    return result;
}

//---------------------------------------------------------------------------//
} // namespace demo_interactor
