#include "hip/hip_runtime.h"
//----------------------------------*-C++-*----------------------------------//
// Copyright 2020 UT-Battelle, LLC, and other Celeritas developers.
// See the top-level COPYRIGHT file for details.
// SPDX-License-Identifier: (Apache-2.0 OR MIT)
//---------------------------------------------------------------------------//
//! \file Utils.cuda.cc
//---------------------------------------------------------------------------//
#include "Utils.hh"

#include "celeritas_config.h"
#if CELERITAS_USE_CUDA
#    include <hip/hip_runtime_api.h>
#endif
#include "base/Assert.hh"

#include <alpaka/alpaka.hpp>

namespace celeritas
{
//---------------------------------------------------------------------------//
// Initialize device in a round-robin fashion from a communicator
void initialize_device(const Communicator& comm)
{
#if CELERITAS_USE_CUDA
    // Get number of devices    
    using Dim = alpaka::dim::DimInt<1>;
    using Idx = uint32_t;
    using Acc = alpaka::acc::AccGpuCudaRt<Dim, Idx>;
    std::size_t num_devices = alpaka::pltf::getDevCount<Acc>();    
    CHECK(num_devices > 0);

    // Set device based on communicator
    //Alpaka has no equivalant of hipSetDevice - they seem to claim it is not needed.
    //int device_id = comm.rank() % num_devices;
    //CELER_CUDA_CALL(hipSetDevice(device_id));
#else
    (void)sizeof(comm);
#endif
}

//---------------------------------------------------------------------------//
} // namespace celeritas
